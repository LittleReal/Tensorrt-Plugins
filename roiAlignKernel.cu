#include "hip/hip_runtime.h"
#include "./roiAlignKernel.h"

#include "NvInfer.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_1D_KERNEL_LOOP(i, n)                                 \
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
        i += blockDim.x * gridDim.x)


// The maximum number of blocks to use in the default kernel call. This value is copied from mxnet
constexpr int ROI_MAXIMUM_NUM_BLOCKS = 4096;

// const int kMemUnitBits = 4; The original defination in mxnet is:
//   #if MSHADOW_OLD_CUDA
//   const int kMemUnitBits = 4;
//   const int kMaxThreadsPerBlock = 512;
//   #else
//   const int kMemUnitBits = 5;
//   const int kMaxThreadsPerBlock = 1024;
//   #endif
const int kMaxThreadsPerBlock = 512;
inline int ROI_GET_BLOCKS(const int N) {
    return std::max(
        std::min(
            (N + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock,
            ROI_MAXIMUM_NUM_BLOCKS),
        // Use at least 1 block, since CUDA does not allow empty block
        1);
}


template <typename T>
__device__ T bilinear_interpolate(
    const T* bottom_data,
    const int height,
    const int width,
    T y,
    T x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  int y_low = static_cast<int>(y);
  int x_low = static_cast<int>(x);
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void ROIAlignForwardKernel(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const bool position_sensitive,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;
    // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    int c_unpooled = c;
    int channels_unpooled = channels;
    if (position_sensitive) {
      c_unpooled = c * pooled_height * pooled_width + ph * pooled_width + pw;
      channels_unpooled = channels * pooled_height * pooled_width;
    }
    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels_unpooled + c_unpooled)
        * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height);  // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w;  // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) {  // e.g., iy = 0, 1
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h);  // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;
    top_data[index] = output_val;
  }
}

using namespace std;
int roiAlignInference(
    hipStream_t stream,
    void **outputs,
    const void* const* inputs,
    const int in_width,
    const int in_height,
    const int in_depth,
    const float spatial_scale,
    const int sample_ratio,
    const int pooled_height,
    const int pooled_width,
    const bool position_sensitive,
    const int n_rois){
    const int count=n_rois*in_depth*pooled_height*pooled_width;
    const float* in_data=static_cast<const float*>(inputs[0]);
    const float* roi_data=static_cast<const float*>(inputs[1]);
    float* out_data=static_cast<float*>(outputs[0]);

    ROIAlignForwardKernel<float><<<count, kMaxThreadsPerBlock, 0, stream>>>(
        count,
        in_data,
        spatial_scale,
        position_sensitive,
        in_depth,
        in_height,
        in_width,
        pooled_height,
        pooled_width,
        sample_ratio,
        roi_data,
        out_data);
    return hipGetLastError() != hipSuccess;
}

